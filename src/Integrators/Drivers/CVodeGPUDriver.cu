#include "hip/hip_runtime.h"
#include <memory>
#include <stdexcept>
#include <cmath>
#include <sstream>

#include "hip/hip_runtime.h"

#include "ODEIntegrator/Integrators/Drivers/CVodeGPUDriver.cuh"
#include "ODEIntegrator/Context/Context.hpp"
#include "ODEIntegrator/Integrators/CVodeGPUDataModels.hpp"

#include "Mechanism/GPU/gpu_memory.cuh"
#include "Mechanism/GPU/dydt.cuh"
#include "Mechanism/GPU/jacob.cuh"

// CUDA block size
#define BLOCKSIZE 32
// Max GPU memory allocation by PyJac
#define MAX_GPU_MEM_PYJAC 0.8

__global__ void kernel_dydt(const int nSystems, const double t, const double P, double *ySun, double* dySun, double *yPy, double *dyPy, mechanism_memory pyjac_mem);
__device__ void sun_to_pyjac_Y(double *ySun, double *yPy);
__device__ void pyjac_to_sun_Y(double *yPy, double *ySun);
__global__ void kernel_eval_jacob(const int nSystems, const double t, const double P, double *ySun, double *JSun, double *yPy, double *JPy, mechanism_memory pyjac_mem);
__device__ void sun_to_pyjac_YJ(double *ySun, double *yPy, double *JSun, double *JPy);
__device__ void pyjac_to_sun_YJ(double *yPy, double *ySun, double *JPy, double *JSun);

int dydt_cvode_GPU(realtype t, N_Vector y, N_Vector ydot, void* userData)
{
    GPUUserData* uData = static_cast<GPUUserData*>(userData);
    auto logger = uData->ctx->getLogger();
    realtype *yptr, *ydotptr, *yptrPy, *ydotptrPy;

    yptr = N_VGetDeviceArrayPointer(y);
    ydotptr = N_VGetDeviceArrayPointer(ydot);

    yptrPy = uData->pyjac_mem->y;
    ydotptrPy = uData->pyjac_mem->dy;

    mechanism_memory pyjac_mem = *(uData->pyjac_mem);

    // Each GPU thread process 1 dydt system
    size_t nBlocks = (int) ceil( ((float) uData->nSystems) / BLOCKSIZE );
    dim3 dimGrid ( nBlocks );
    dim3 dimBlock ( BLOCKSIZE );

    /* Kernel Call */
    kernel_dydt<<< dimGrid, dimBlock >>>(uData->nSystems, t, uData->pressure, yptr, ydotptr, yptrPy, ydotptrPy, pyjac_mem);
    hipDeviceSynchronize();

    #ifdef TESTING
    //uData->test_y_sun_vs_py->ysun_vs_ypyjac();
    //uData->test_y_sun_vs_py->ysun_vs_dypyjac();
    #endif

    hipError_t cudaErr = hipGetLastError();
    if (cudaErr != hipSuccess) {
        std::stringstream ss;
        ss << "hipGetLastError returned: " << hipGetErrorName(cudaErr);
        logger->error(ss.str());
        return -1;
    }

    return 0;
}

int jacobian_cvode_GPU(double t, N_Vector y, N_Vector ydot, SUNMatrix J, void* userData, N_Vector tmp1, N_Vector tmp2, N_Vector tmp3)
{
    GPUUserData* uData = static_cast<GPUUserData*>(userData);
    auto logger = uData->ctx->getLogger();
    realtype *Jptr, *yptr, *JptrPy, *yptrPy;

    Jptr = SUNMatrix_MagmaDense_Data(J);
    yptr = N_VGetDeviceArrayPointer_Cuda(y);

    /* Length of batched Jacobian matrix (should be = NSP * NSP * gpu_points) */
    // sunindextype JLength = SUNMatrix_MagmaDense_LData(J);

    JptrPy = uData->pyjac_mem->jac;
    yptrPy = uData->pyjac_mem->y;

    mechanism_memory pyjac_mem = *(uData->pyjac_mem);

    // Each GPU thread evaluate 1 jacobian matrix (1 system per thread)
    size_t nBlocks = (int) ceil( ((float) uData->nSystems) / BLOCKSIZE );
    dim3 dimGrid ( nBlocks );
    dim3 dimBlock ( BLOCKSIZE );

    /* Kernel call */
    kernel_eval_jacob<<< dimGrid, dimBlock >>>(uData->nSystems, t, uData->pressure, yptr, Jptr, yptrPy, JptrPy, pyjac_mem);

    #ifdef TESTING
    uData->test_jacobian->compare_matrices();
    #endif

    hipDeviceSynchronize();
    hipError_t cudaErr = hipGetLastError();
    if (cudaErr != hipSuccess) {
        std::stringstream ss;
        ss << "hipGetLastError returned: " << hipGetErrorName(cudaErr);
        logger->error(ss.str());
        return -1;
    }

    return 0;
}

int calc_gpu_points(std::shared_ptr<Context> ctx, int total_points, int &real_calculated_points) {
    auto logger = ctx->getLogger();
    size_t mech_size = required_mechanism_size();
    size_t free_mem = 0;
    size_t total_mem = 0;

    cudaErrorCheck( hipMemGetInfo(&free_mem, &total_mem) );

    int max_allocated_points = int(floor( MAX_GPU_MEM_PYJAC * ((double)free_mem / (double)mech_size) ));

    // Choose between the remaining points and the maximum allocatable 
    real_calculated_points = std::min(total_points, max_allocated_points);

    // Transform padded in a number multiple of BLOCKSIZE, ej: 1000 -> 1024
    int padded = int(std::ceil(real_calculated_points / float(BLOCKSIZE)) * BLOCKSIZE);

    if (padded == 0) {
        logger->error("Mechanism is too large, cannot allocate any point on GPU");
        throw std::runtime_error("Mechanism is too large, cannot allocate any point on GPU");
    }

    return padded;
}

void init_memory_gpu(std::shared_ptr<Context> ctx, int num_systems, mechanism_memory *pyjac_mem) {
    auto logger = ctx->getLogger();
    logger->info("Initializing PyJac GPU memory...");

    hipError_t cuda_err;

    /* Allocate Memory */

    cuda_err = hipMalloc((void **) &(pyjac_mem->y), NSP * num_systems * sizeof(double));
    if (cuda_err != hipSuccess) {
        logger->error("hipMalloc error code: " + cuda_err);
    }

    cuda_err = hipMalloc((void **) &(pyjac_mem->dy), NSP * num_systems * sizeof(double));
    if (cuda_err != hipSuccess) {
        logger->error("hipMalloc error code: " + cuda_err);
    }

    cuda_err = hipMalloc((void **) &(pyjac_mem->conc), NSP * num_systems * sizeof(double));
    if (cuda_err != hipSuccess) {
        logger->error("hipMalloc error code: " + cuda_err);
    }

    cuda_err = hipMalloc((void **) &(pyjac_mem->fwd_rates), FWD_RATES * num_systems * sizeof(double));
    if (cuda_err != hipSuccess) {
        logger->error("hipMalloc error code: " + cuda_err);
    }

    cuda_err = hipMalloc((void **) &(pyjac_mem->rev_rates), REV_RATES * num_systems * sizeof(double));
    if (cuda_err != hipSuccess) {
        logger->error("hipMalloc error code: " + cuda_err);
    }

    cuda_err = hipMalloc((void **) &(pyjac_mem->spec_rates), NSP * num_systems * sizeof(double));
    if (cuda_err != hipSuccess) {
        logger->error("hipMalloc error code: " + cuda_err);
    }

    cuda_err = hipMalloc((void **) &(pyjac_mem->cp), NSP * num_systems * sizeof(double));
    if (cuda_err != hipSuccess) {
        logger->error("hipMalloc error code: " + cuda_err);
    }

    cuda_err = hipMalloc((void **) &(pyjac_mem->h), NSP * num_systems * sizeof(double));
    if (cuda_err != hipSuccess) {
        logger->error("hipMalloc error code: " + cuda_err);
    }

    cuda_err = hipMalloc((void **) &(pyjac_mem->dBdT), NSP * num_systems * sizeof(double));
    if (cuda_err != hipSuccess) {
        logger->error("hipMalloc error code: " + cuda_err);
    }

    cuda_err = hipMalloc((void **) &(pyjac_mem->jac), NSP * NSP * num_systems * sizeof(double));
    if (cuda_err != hipSuccess) {
        logger->error("hipMalloc error code: " + cuda_err);
    }

    cuda_err = hipMalloc((void **) &(pyjac_mem->var), 1 * num_systems * sizeof(double));
    if (cuda_err != hipSuccess) {
        logger->error("hipMalloc error code: " + cuda_err);
    }

    cuda_err = hipMalloc((void **) &(pyjac_mem->J_nplusjplus), NSP * num_systems * sizeof(double));
    if (cuda_err != hipSuccess) {
        logger->error("hipMalloc error code: " + cuda_err);
    }

    cuda_err = hipMalloc((void **) &(pyjac_mem->pres_mod), PRES_MOD_RATES * num_systems * sizeof(double));
    if (cuda_err != hipSuccess) {
        logger->error("hipMalloc error code: " + cuda_err);
    }

    /* Init memory */

    cuda_err = hipMemset(pyjac_mem->spec_rates, 0, NSP * num_systems * sizeof(double));
    if (cuda_err != hipSuccess) {
        logger->error("hipMemset error code: " + cuda_err);
    }

    cuda_err = hipMemset(pyjac_mem->dy, 0, NSP * num_systems * sizeof(double));
    if (cuda_err != hipSuccess) {
        logger->error("hipMemset error code: " + cuda_err);
    }

    cuda_err = hipMemset(pyjac_mem->jac, 0, NSP * NSP * num_systems * sizeof(double));
    if (cuda_err != hipSuccess) {
        logger->error("hipMemset error code: " + cuda_err);
    }
    std::stringstream ss;
    ss << "GPU systems allocated in this iteration: " << num_systems;
    logger->info(ss.str());
    logger->info("PyJac GPU memory initialized");
}

void free_memory_gpu(std::shared_ptr<Context> ctx, mechanism_memory *pyjac_mem) {
    auto logger = ctx->getLogger();
    hipError_t cuda_err;

    cuda_err = hipFree(pyjac_mem->y);
    if (cuda_err != hipSuccess) {
        logger->error("hipFree error code: " + cuda_err);
    }

    cuda_err = hipFree(pyjac_mem->dy);
    if (cuda_err != hipSuccess) {
        logger->error("hipFree error code: " + cuda_err);
    }

    cuda_err = hipFree(pyjac_mem->conc);
    if (cuda_err != hipSuccess) {
        logger->error("hipFree error code: " + cuda_err);
    }

    cuda_err = hipFree(pyjac_mem->fwd_rates);
    if (cuda_err != hipSuccess) {
        logger->error("hipFree error code: " + cuda_err);
    }

    cuda_err = hipFree(pyjac_mem->rev_rates);
    if (cuda_err != hipSuccess) {
        logger->error("hipFree error code: " + cuda_err);
    }

    cuda_err = hipFree(pyjac_mem->spec_rates);
    if (cuda_err != hipSuccess) {
        logger->error("hipFree error code: " + cuda_err);
    }

    cuda_err = hipFree(pyjac_mem->cp);
    if (cuda_err != hipSuccess) {
        logger->error("hipFree error code: " + cuda_err);
    }

    cuda_err = hipFree(pyjac_mem->h);
    if (cuda_err != hipSuccess) {
        logger->error("hipFree error code: " + cuda_err);
    }

    cuda_err = hipFree(pyjac_mem->dBdT);
    if (cuda_err != hipSuccess) {
        logger->error("hipFree error code: " + cuda_err);
    }

    cuda_err = hipFree(pyjac_mem->jac);
    if (cuda_err != hipSuccess) {
        logger->error("hipFree error code: " + cuda_err);
    }

    cuda_err = hipFree(pyjac_mem->var);
    if (cuda_err != hipSuccess) {
        logger->error("hipFree error code: " + cuda_err);
    }

    cuda_err = hipFree(pyjac_mem->J_nplusjplus);
    if (cuda_err != hipSuccess) {
        logger->error("hipFree error code: " + cuda_err);
    }

    cuda_err = hipFree(pyjac_mem->pres_mod);
    if (cuda_err != hipSuccess) {
        logger->error("hipFree error code: " + cuda_err);
    }

    logger->info("PyJac GPU memory freed");
}

__global__ void kernel_dydt(const int nSystems, const double t, const double P, double *ySun, double* dySun,
                            double *yPy, double *dyPy, mechanism_memory pyjac_mem) {
    if (T_ID < nSystems) {
        // Give the device pointers to a device structure pointer
        mechanism_memory *d_mem = &pyjac_mem;

        // Reorder data for PyJac
        sun_to_pyjac_Y(ySun, yPy);

        dydt(t, P, yPy, dyPy, d_mem);

        // Reorder data back to Sundials
        pyjac_to_sun_Y(dyPy, dySun);
    }
}

__device__ void sun_to_pyjac_Y(double *ySun, double *yPy) {

    int threadID = threadIdx.x + blockIdx.x * blockDim.x;

    // PyJac index -> #define INDEX(i) (T_ID + (i) * GRID_DIM)

    // ySun = {T0, Y00, Y01, ... Y0(NSP-1), T1, Y10, Y11, ... Y1(NSP-1), ...}
    // yPy  = {T0,  T1,  T2, ...  T(nSystems), Y00, Y10, Y20, ..., Y(nSystems)0, Y01, Y11, Y21, ..., Y(nSystems)1, ...}

    for (int i = 0; i < NSP; i++) {
        yPy[INDEX(i)] = ySun[threadID * NSP + i];
    }
}

__device__ void pyjac_to_sun_Y(double *yPy, double *ySun) {

    int threadID = threadIdx.x + blockIdx.x * blockDim.x;

    for (int i = 0; i < NSP; i++) {
        ySun[threadID * NSP + i] = yPy[INDEX(i)];
    }
}

__global__ void kernel_eval_jacob(const int nSystems, const double t, const double P, double *ySun, double *JSun,
                                  double *yPy, double *JPy, mechanism_memory pyjac_mem) {

    if (T_ID < nSystems) {
        // Give the device pointers to a device structure pointer
        mechanism_memory *d_mem = &pyjac_mem;

        // Reorder data for PyJac
        sun_to_pyjac_YJ(ySun, yPy, JSun, JPy);

        // Jacobian analytic evaluation with PyJac
        eval_jacob(t, P, yPy, JPy, d_mem);

        // Reorder data back to Sundials
        pyjac_to_sun_YJ(yPy, ySun, JPy, JSun);
    }
}

__device__ void sun_to_pyjac_YJ(double *ySun, double *yPy, double *JSun, double *JPy) {

    int threadID = threadIdx.x + blockIdx.x * blockDim.x;

    // PyJac index -> #define INDEX(i) (T_ID + (i) * GRID_DIM)

    // ySun = {T0, Y00, Y01, ... Y0(NSP-1), T1, Y10, Y11, ... Y1(NSP-1), ...}
    // yPy  = {T0,  T1,  T2, ...  T(nSystems), Y00, Y10, Y20, ..., Y(nSystems)0, Y01, Y11, Y21, ..., Y(nSystems)1, ...}

    // JSun = {J0, J1, ..., J(nSystems)} Each matrix is ordered column-major
    // JPy  = same structure that yPy, ordered column major

    // Location of the first system element for the current thread for "y" vector
    int sunSystemY = threadID * NSP;

    // Location of the first element of the system Jacobian matrix
    int sunSystemJac = threadID * NSP * NSP;

    for (int j = 0; j < NSP; j++) {
        yPy[INDEX(j)] = ySun[sunSystemY + j];
        for (int i = 0; i < NSP; i++) {
            JPy[INDEX(j*NSP + i)] =  JSun[sunSystemJac + j*NSP + i];
        }
    }
}

__device__ void pyjac_to_sun_YJ(double *yPy, double *ySun, double *JPy, double *JSun) {

    int threadID = threadIdx.x + blockIdx.x * blockDim.x;

    // Location of the first system element for the current thread for "y" vector
    int sunSystemY = threadID * NSP;

    // Location of the first element of the system Jacobian matrix
    int sunSystemJac = threadID * NSP * NSP;

    for (int j = 0; j < NSP; j++) {
        ySun[sunSystemY + j] = yPy[INDEX(j)];
        for (int i = 0; i < NSP; i++) {
            JSun[sunSystemJac + j*NSP + i] = JPy[INDEX(j * NSP + i)];
        }
    }
}

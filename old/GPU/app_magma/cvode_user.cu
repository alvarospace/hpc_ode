#include "hip/hip_runtime.h"
#include <app_magma/cvode_user.cuh>

int eval_jacob_cvode(double t, N_Vector y, N_Vector ydot, SUNMatrix J, void* userData, N_Vector tmp1, N_Vector tmp2, N_Vector tmp3)
{
  UserData *uData = (UserData*) userData;
  realtype *Jptr, *yptr, *JptrPy, *yptrPy;

  Jptr = SUNMatrix_MagmaDense_Data(J);
  yptr = N_VGetDeviceArrayPointer_Cuda(y);

  /* Length of batched Jacobian matrix (should be = NSP * NSP * gpu_points) */
  // sunindextype JLength = SUNMatrix_MagmaDense_LData(J);

  JptrPy = uData->pyjac_mem->jac;
  yptrPy = uData->pyjac_mem->y;

  mechanism_memory pyjac_mem = *(uData->pyjac_mem);
  
  // Each GPU thread evaluate 1 jacobian matrix (1 system per thread)
  size_t nBlocks = (int) ceil( ((float) uData->nSystems) / BLOCKSIZE );
  dim3 dimGrid ( nBlocks );
  dim3 dimBlock ( BLOCKSIZE );

  /* Kernel call */
  kernel_eval_jacob<<< dimGrid, dimBlock >>>(uData->nSystems, t, uData->Pressure, yptr, Jptr, yptrPy, JptrPy, pyjac_mem);
  
  #ifdef TESTING
  uData->test_jacobian->compare_matrices();
  #endif

  hipDeviceSynchronize();
  hipError_t cudaErr = hipGetLastError();
  if (cudaErr != hipSuccess) {
    fprintf(stderr, "\t ERROR in 'dydt_cvode': hipGetLastError returned %s", hipGetErrorName(cudaErr));
    return -1;
  }

	return 0;
}

int dydt_cvode(realtype t, N_Vector y, N_Vector ydot, void* userData)
{
  UserData *uData = (UserData*) userData;
  realtype *yptr, *ydotptr, *yptrPy, *ydotptrPy;

  yptr = N_VGetDeviceArrayPointer(y);
  ydotptr = N_VGetDeviceArrayPointer(ydot);

  yptrPy = uData->pyjac_mem->y;
  ydotptrPy = uData->pyjac_mem->dy;

  mechanism_memory pyjac_mem = *(uData->pyjac_mem);

  // Each GPU thread evaluate 1 dydt system
  size_t nBlocks = (int) ceil( ((float) uData->nSystems) / BLOCKSIZE );
  dim3 dimGrid ( nBlocks );
  dim3 dimBlock ( BLOCKSIZE );

  /* Kernel Call */
  kernel_dydt<<< dimGrid, dimBlock >>>(uData->nSystems, t, uData->Pressure, yptr, ydotptr, yptrPy, ydotptrPy, pyjac_mem);
  hipDeviceSynchronize();

  #ifdef TESTING
  //uData->test_y_sun_vs_py->ysun_vs_ypyjac();
  //uData->test_y_sun_vs_py->ysun_vs_dypyjac();
  #endif

  hipError_t cudaErr = hipGetLastError();
  if (cudaErr != hipSuccess) {
    fprintf(stderr, "\t ERROR in 'dydt_cvode': hipGetLastError returned %s", hipGetErrorName(cudaErr));
    return -1;
  }

  return 0;
}

__global__ void kernel_dydt(const int nSystems, const double t, const double P, double *ySun, double* dySun,
                            double *yPy, double *dyPy, mechanism_memory pyjac_mem) {
  if (T_ID < nSystems) {
    // Give the device pointers to a device structure pointer
    mechanism_memory *d_mem = &pyjac_mem;

    // Reorder data for PyJac
    sun_to_pyjac_Y(ySun, yPy);

    dydt(t, P, yPy, dyPy, d_mem);

    // Reorder data back to Sundials
    pyjac_to_sun_Y(dyPy, dySun);
  }
}

__global__ void kernel_eval_jacob(const int nSystems, const double t, const double P, double *ySun, double *JSun,
                                  double *yPy, double *JPy, mechanism_memory pyjac_mem) {

  if (T_ID < nSystems) {
    // Give the device pointers to a device structure pointer
    mechanism_memory *d_mem = &pyjac_mem;

    // Reorder data for PyJac
    sun_to_pyjac_YJ(ySun, yPy, JSun, JPy);

    // Jacobian analytic evaluation with PyJac
    eval_jacob(t, P, yPy, JPy, d_mem);

    // Reorder data back to Sundials
    pyjac_to_sun_YJ(yPy, ySun, JPy, JSun);
  }
}

__device__ void sun_to_pyjac_YJ(double *ySun, double *yPy, double *JSun, double *JPy) {

  int threadID = threadIdx.x + blockIdx.x * blockDim.x;

  // PyJac index -> #define INDEX(i) (T_ID + (i) * GRID_DIM)

  // ySun = {T0, Y00, Y01, ... Y0(NSP-1), T1, Y10, Y11, ... Y1(NSP-1), ...}
  // yPy  = {T0,  T1,  T2, ...  T(nSystems), Y00, Y10, Y20, ..., Y(nSystems)0, Y01, Y11, Y21, ..., Y(nSystems)1, ...}

  // JSun = {J0, J1, ..., J(nSystems)} Each matrix is ordered column-major
  // JPy  = same structure that yPy, ordered column major

  // Location of the first system element for the current thread for "y" vector
  int sunSystemY = threadID * NSP;

  // Location of the first element of the system Jacobian matrix
  int sunSystemJac = threadID * NSP * NSP;

  for (int j = 0; j < NSP; j++) {
    yPy[INDEX(j)] = ySun[sunSystemY + j];
    for (int i = 0; i < NSP; i++) {
      JPy[INDEX(j*NSP + i)] =  JSun[sunSystemJac + j*NSP + i];
    }
  }
}

__device__ void pyjac_to_sun_YJ(double *yPy, double *ySun, double *JPy, double *JSun) {

  int threadID = threadIdx.x + blockIdx.x * blockDim.x;

  // Location of the first system element for the current thread for "y" vector
  int sunSystemY = threadID * NSP;

  // Location of the first element of the system Jacobian matrix
  int sunSystemJac = threadID * NSP * NSP;

  for (int j = 0; j < NSP; j++) {
    ySun[sunSystemY + j] = yPy[INDEX(j)];
    for (int i = 0; i < NSP; i++) {
      JSun[sunSystemJac + j*NSP + i] = JPy[INDEX(j * NSP + i)];
    }
  }

}

__device__ void sun_to_pyjac_Y(double *ySun, double *yPy) {

  int threadID = threadIdx.x + blockIdx.x * blockDim.x;

  // PyJac index -> #define INDEX(i) (T_ID + (i) * GRID_DIM)

  // ySun = {T0, Y00, Y01, ... Y0(NSP-1), T1, Y10, Y11, ... Y1(NSP-1), ...}
  // yPy  = {T0,  T1,  T2, ...  T(nSystems), Y00, Y10, Y20, ..., Y(nSystems)0, Y01, Y11, Y21, ..., Y(nSystems)1, ...}

  for (int i = 0; i < NSP; i++) {
    yPy[INDEX(i)] = ySun[threadID * NSP + i];
  }
}

__device__ void pyjac_to_sun_Y(double *yPy, double *ySun) {

  int threadID = threadIdx.x + blockIdx.x * blockDim.x;

  for (int i = 0; i < NSP; i++) {
    ySun[threadID * NSP + i] = yPy[INDEX(i)];
  }
}

int calc_gpu_points(int total_points, int &real_calculated_points) {
    size_t mech_size = required_mechanism_size();
    size_t free_mem = 0;
    size_t total_mem = 0;

    cudaErrorCheck( hipMemGetInfo(&free_mem, &total_mem) );

    int max_allocated_points = int(floor( MAX_GPU_MEM_PYJAC * ((double)free_mem / (double)mech_size) ));

    // Choose between the remaining points and the maximum allocatable 
    real_calculated_points = min(total_points, max_allocated_points);

    // Transform padded in a number multiple of BLOCKSIZE, ej: 1000 -> 1024
    int padded = int(ceil(real_calculated_points / float(BLOCKSIZE)) * BLOCKSIZE);

    if (padded == 0) {
        std::cout << "Mechanism is too large, cannot allocate any point... exiting program." << std::endl;
        exit(EXIT_FAILURE);
    }

    std::cout << "Initializing PyJac GPU memory..." << std::endl;
    std::cout << "GPU allocated points in this iteration: " << padded << std::endl;
    return padded;
}

void init_memory_gpu(int num_systems, mechanism_memory *pyjac_mem) {
    hipError_t cuda_err;

    /* Allocate Memory */

    cuda_err = hipMalloc((void **) &(pyjac_mem->y), NSP * num_systems * sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cout << "Error: hipMalloc error code " << cuda_err << " on function \"";
        std::cout << __func__ << "\" (line: " << __LINE__ << ")" << std::endl;
    }

    cuda_err = hipMalloc((void **) &(pyjac_mem->dy), NSP * num_systems * sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cout << "Error: hipMalloc error code " << cuda_err << " on function \"";
        std::cout << __func__ << "\" (line: " << __LINE__ << ")" << std::endl;
    }

    cuda_err = hipMalloc((void **) &(pyjac_mem->conc), NSP * num_systems * sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cout << "Error: hipMalloc error code " << cuda_err << " on function \"";
        std::cout << __func__ << "\" (line: " << __LINE__ << ")" << std::endl;
    }

    cuda_err = hipMalloc((void **) &(pyjac_mem->fwd_rates), FWD_RATES * num_systems * sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cout << "Error: hipMalloc error code " << cuda_err << " on function \"";
        std::cout << __func__ << "\" (line: " << __LINE__ << ")" << std::endl;
    }

    cuda_err = hipMalloc((void **) &(pyjac_mem->rev_rates), REV_RATES * num_systems * sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cout << "Error: hipMalloc error code " << cuda_err << " on function \"";
        std::cout << __func__ << "\" (line: " << __LINE__ << ")" << std::endl;
    }

    cuda_err = hipMalloc((void **) &(pyjac_mem->spec_rates), NSP * num_systems * sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cout << "Error: hipMalloc error code " << cuda_err << " on function \"";
        std::cout << __func__ << "\" (line: " << __LINE__ << ")" << std::endl;
    }

    cuda_err = hipMalloc((void **) &(pyjac_mem->cp), NSP * num_systems * sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cout << "Error: hipMalloc error code " << cuda_err << " on function \"";
        std::cout << __func__ << "\" (line: " << __LINE__ << ")" << std::endl;
    }

    cuda_err = hipMalloc((void **) &(pyjac_mem->h), NSP * num_systems * sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cout << "Error: hipMalloc error code " << cuda_err << " on function \"";
        std::cout << __func__ << "\" (line: " << __LINE__ << ")" << std::endl;
    }

    cuda_err = hipMalloc((void **) &(pyjac_mem->dBdT), NSP * num_systems * sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cout << "Error: hipMalloc error code " << cuda_err << " on function \"";
        std::cout << __func__ << "\" (line: " << __LINE__ << ")" << std::endl;
    }

    cuda_err = hipMalloc((void **) &(pyjac_mem->jac), NSP * NSP * num_systems * sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cout << "Error: hipMalloc error code " << cuda_err << " on function \"";
        std::cout << __func__ << "\" (line: " << __LINE__ << ")" << std::endl;
    }

    cuda_err = hipMalloc((void **) &(pyjac_mem->var), 1 * num_systems * sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cout << "Error: hipMalloc error code " << cuda_err << " on function \"";
        std::cout << __func__ << "\" (line: " << __LINE__ << ")" << std::endl;
    }

    cuda_err = hipMalloc((void **) &(pyjac_mem->J_nplusjplus), NSP * num_systems * sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cout << "Error: hipMalloc error code " << cuda_err << " on function \"";
        std::cout << __func__ << "\" (line: " << __LINE__ << ")" << std::endl;
    }

    cuda_err = hipMalloc((void **) &(pyjac_mem->pres_mod), PRES_MOD_RATES * num_systems * sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cout << "Error: hipMalloc error code " << cuda_err << " on function \"";
        std::cout << __func__ << "\" (line: " << __LINE__ << ")" << std::endl;
    }

    /* Init memory */

    cuda_err = hipMemset(pyjac_mem->spec_rates, 0, NSP * num_systems * sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cout << "Error: hipMemset error code " << cuda_err << " on function \"";
        std::cout << __func__ << "\" (line: " << __LINE__ << ")" << std::endl;
    }

    cuda_err = hipMemset(pyjac_mem->dy, 0, NSP * num_systems * sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cout << "Error: hipMemset error code " << cuda_err << " on function \"";
        std::cout << __func__ << "\" (line: " << __LINE__ << ")" << std::endl;
    }

    cuda_err = hipMemset(pyjac_mem->jac, 0, NSP * NSP * num_systems * sizeof(double));
    if (cuda_err != hipSuccess) {
        std::cout << "Error: hipMemset error code " << cuda_err << " on function \"";
        std::cout << __func__ << "\" (line: " << __LINE__ << ")" << std::endl;
    }
}

void free_memory_gpu(mechanism_memory *pyjac_mem) {
    hipError_t cuda_err;

    cuda_err = hipFree(pyjac_mem->y);
    if (cuda_err != hipSuccess) {
        std::cout << "Error: hipFree error code " << cuda_err << " on function \"";
        std::cout << __func__ << "\" (line: " << __LINE__ << ")" << std::endl;
    }

    cuda_err = hipFree(pyjac_mem->dy);
    if (cuda_err != hipSuccess) {
        std::cout << "Error: hipFree error code " << cuda_err << " on function \"";
        std::cout << __func__ << "\" (line: " << __LINE__ << ")" << std::endl;
    }

    cuda_err = hipFree(pyjac_mem->conc);
    if (cuda_err != hipSuccess) {
        std::cout << "Error: hipFree error code " << cuda_err << " on function \"";
        std::cout << __func__ << "\" (line: " << __LINE__ << ")" << std::endl;
    }

    cuda_err = hipFree(pyjac_mem->fwd_rates);
    if (cuda_err != hipSuccess) {
        std::cout << "Error: hipFree error code " << cuda_err << " on function \"";
        std::cout << __func__ << "\" (line: " << __LINE__ << ")" << std::endl;
    }

    cuda_err = hipFree(pyjac_mem->rev_rates);
    if (cuda_err != hipSuccess) {
        std::cout << "Error: hipFree error code " << cuda_err << " on function \"";
        std::cout << __func__ << "\" (line: " << __LINE__ << ")" << std::endl;
    }

    cuda_err = hipFree(pyjac_mem->spec_rates);
    if (cuda_err != hipSuccess) {
        std::cout << "Error: hipFree error code " << cuda_err << " on function \"";
        std::cout << __func__ << "\" (line: " << __LINE__ << ")" << std::endl;
    }

    cuda_err = hipFree(pyjac_mem->cp);
    if (cuda_err != hipSuccess) {
        std::cout << "Error: hipFree error code " << cuda_err << " on function \"";
        std::cout << __func__ << "\" (line: " << __LINE__ << ")" << std::endl;
    }

    cuda_err = hipFree(pyjac_mem->h);
    if (cuda_err != hipSuccess) {
        std::cout << "Error: hipFree error code " << cuda_err << " on function \"";
        std::cout << __func__ << "\" (line: " << __LINE__ << ")" << std::endl;
    }

    cuda_err = hipFree(pyjac_mem->dBdT);
    if (cuda_err != hipSuccess) {
        std::cout << "Error: hipFree error code " << cuda_err << " on function \"";
        std::cout << __func__ << "\" (line: " << __LINE__ << ")" << std::endl;
    }

    cuda_err = hipFree(pyjac_mem->jac);
    if (cuda_err != hipSuccess) {
        std::cout << "Error: hipFree error code " << cuda_err << " on function \"";
        std::cout << __func__ << "\" (line: " << __LINE__ << ")" << std::endl;
    }

    cuda_err = hipFree(pyjac_mem->var);
    if (cuda_err != hipSuccess) {
        std::cout << "Error: hipFree error code " << cuda_err << " on function \"";
        std::cout << __func__ << "\" (line: " << __LINE__ << ")" << std::endl;
    }

    cuda_err = hipFree(pyjac_mem->J_nplusjplus);
    if (cuda_err != hipSuccess) {
        std::cout << "Error: hipFree error code " << cuda_err << " on function \"";
        std::cout << __func__ << "\" (line: " << __LINE__ << ")" << std::endl;
    }

    cuda_err = hipFree(pyjac_mem->pres_mod);
    if (cuda_err != hipSuccess) {
        std::cout << "Error: hipFree error code " << cuda_err << " on function \"";
        std::cout << __func__ << "\" (line: " << __LINE__ << ")" << std::endl;
    }

}



/*
 * Check function return value...
 *   opt == 0 means SUNDIALS function allocates memory so check if
 *            returned NULL pointer
 *   opt == 1 means SUNDIALS function returns an integer value so check if
 *            retval < 0
 *   opt == 2 means function allocates memory so check if returned
 *            NULL pointer
 */

int check_retval(void *returnvalue, const char *funcname, int opt)
{
  int *retval;

  /* Check if SUNDIALS function returned NULL pointer - no memory allocated */
  if (opt == 0 && returnvalue == NULL) {
    fprintf(stderr, "\nSUNDIALS_ERROR: %s() failed - returned NULL pointer\n\n",
            funcname);
    return(1); }

  /* Check if retval < 0 */
  else if (opt == 1) {
    retval = (int *) returnvalue;
    if (*retval < 0) {
      fprintf(stderr, "\nSUNDIALS_ERROR: %s() failed with retval = %d\n\n",
              funcname, *retval);
      return(1); }}

  /* Check if function returned NULL pointer - no memory allocated */
  else if (opt == 2 && returnvalue == NULL) {
    fprintf(stderr, "\nMEMORY_ERROR: %s() failed - returned NULL pointer\n\n",
            funcname);
    return(1); }

  return(0);
}